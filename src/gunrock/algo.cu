#include "hip/hip_runtime.h"
#include "algo.cuh"

#define NOMINMAX

#include "algo.cuh"

#include <algorithm>
#include <filesystem>
#include <vector>

#include <gunrock/algorithms/tc.hxx>
#include <gunrock/formats/csr.hxx>
#include <gunrock/formats/formats.hxx>
#include <gunrock/graph/build.hxx>
#include <gunrock/graph/graph.hxx>
#include <gunrock/io/matrix_market.hxx>
#include <gunrock/memory.hxx>

using vertex_t = uint32_t;
using edge_t = uint32_t;
using weight_t = float;
using graph_t = gunrock::graph::graph_t<
    gunrock::memory::device, unsigned int, unsigned int, float,
    gunrock::graph::graph_csr_t<gunrock::memory::device, unsigned int,
                                unsigned int, float>>;

namespace {
std::optional<
    gunrock::format::csr_t<gunrock::memory::device, vertex_t, edge_t, weight_t>>
    g_csr;
std::optional<graph_t> g_graph;
thrust::device_vector<vertex_t> *g_triangles_count = nullptr;
bool g_initialized = false;
} // namespace

void loadGunrockGraph(std::vector<std::pair<int, int>> edges, int maxRow,
                      int maxCol) {
  using namespace gunrock;

  vertex_t V = std::max(maxRow, maxCol) + 1;

  std::set<std::pair<vertex_t, vertex_t>> unique_edges;

  for (auto [u, v] : edges) {
    if (u != v) {
      unique_edges.insert({u, v});
      unique_edges.insert({v, u});
    }
  }

  int E2 = unique_edges.size();

  std::vector<vertex_t> h_src, h_dst;
  std::vector<weight_t> h_w;

  for (auto [u, v] : unique_edges) {
    h_src.push_back(u);
    h_dst.push_back(v);
    h_w.push_back(1);
  }

  g_csr = format::csr_t<device, vertex_t, edge_t, weight_t>();
  format::coo_t<host, vertex_t, edge_t, weight_t> h_coo{
      V, V, static_cast<uint32_t>(E2)};

  h_coo.row_indices = h_src;
  h_coo.column_indices = h_dst;
  h_coo.nonzero_values = h_w;

  g_csr->from_coo(h_coo);

  gunrock::graph::graph_properties_t graph_properties{false, false, true};
  g_graph = gunrock::graph::build(graph_properties, *g_csr);

  g_triangles_count =
      new thrust::device_vector<vertex_t>(g_graph->get_number_of_vertices(), 0);

  g_initialized = true;
}

size_t runTriangleCountingCuda() {
  if (!g_initialized || !g_graph || !g_triangles_count) {
    throw std::runtime_error(
        "Graph not initialized before running triangle counting");
  }

  bool reduce_all_triangles = true;
  size_t total_triangles = 0;

  gunrock::tc::run(*g_graph, reduce_all_triangles,
                   g_triangles_count->data().get(), &total_triangles);

  return total_triangles / 3;
}

void cleanupGunrockResources() {
  g_graph.reset();
  g_csr.reset();

  if (g_triangles_count) {
    delete g_triangles_count;
    g_triangles_count = nullptr;
  }

  g_initialized = false;
}
